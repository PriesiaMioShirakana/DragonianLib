﻿#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "hip/hip_runtime.h"
#include "kernel.h"

namespace DragonianLib
{
	namespace CudaProvider
	{
		thread_local std::string __LastError = "Empty";  // NOLINT(misc-use-internal-linkage)

		void* cudaAllocate(size_t size)
		{
			void* block = nullptr;
			size_t memFree, memTotal;
			hipMemGetInfo(&memFree, &memTotal);
			if (size > memFree)
			{
				fprintf(stderr, "Cuda memory allocation failed: %zu bytes requested, but only %zu bytes available.\n", size, memFree);
				throw std::runtime_error("Cuda Out of Memory!");
			}
			if (const auto err = ::hipMalloc(&block, size))
				fprintf(stderr, "%s\n", hipGetErrorString(err));
			return block;
		}

		int hipFree(void* block) noexcept
		{
			return ::hipFree(block);
		}

		int host2Device(void* dst, const void* src, size_t size, stream_t stream) noexcept
		{
			return hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, (hipStream_t)stream);
		}

		int device2Host(void* dst, const void* src, size_t size, stream_t stream) noexcept
		{
			return hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, (hipStream_t)stream);
		}

		int device2Device(void* dst, const void* src, size_t size, stream_t stream) noexcept
		{
			return hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToDevice, (hipStream_t)stream);
		}

		const std::string& getLastErrorString() noexcept
		{
			return __LastError;
		}

		stream_t createCudaStream() noexcept
		{
			hipStream_t Ret;
			if (const auto err = hipStreamCreate(&Ret))
				fprintf(stderr, "%s\n", hipGetErrorString(err));
			return stream_t(Ret);
		}

		int destroyCudaStream(stream_t stream) noexcept
		{
			return hipStreamDestroy((hipStream_t)stream);
		}

		int asyncCudaStream(stream_t stream) noexcept
		{
			return hipStreamSynchronize((hipStream_t)stream);
		}

		const char* getCudaError(int errorId) noexcept
		{
			return hipGetErrorString(static_cast<hipError_t>(errorId));
		}

		int getLastError() noexcept
		{
			return hipGetLastError();
		}
	}
}
