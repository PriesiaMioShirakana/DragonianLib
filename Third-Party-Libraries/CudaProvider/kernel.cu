﻿#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "hip/hip_runtime.h"
#include "kernel.h"

namespace DragonianLib
{
	namespace CudaProvider
	{
		thread_local std::string __LastError = "Empty";  // NOLINT(misc-use-internal-linkage)

		void* cudaAllocate(size_t size)
		{
			void* block = nullptr;
			if (auto err = ::hipMalloc(&block, size))
				fprintf(stderr, "%s\n", hipGetErrorString(err));
			return block;
		}

		int hipFree(void* block)
		{
			return ::hipFree(block);
		}

		int host2Device(void* dst, const void* src, size_t size, stream_t stream)
		{
			return hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, (hipStream_t)stream);
		}

		int device2Host(void* dst, const void* src, size_t size, stream_t stream)
		{
			return hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, (hipStream_t)stream);
		}

		int device2Device(void* dst, const void* src, size_t size, stream_t stream)
		{
			return hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToDevice, (hipStream_t)stream);
		}

		const std::string& getLastErrorString()
		{
			return __LastError;
		}

		stream_t createCudaStream()
		{
			hipStream_t Ret;
			if (auto err = hipStreamCreate(&Ret))
				fprintf(stderr, "%s\n", hipGetErrorString(err));
			return stream_t(Ret);
		}

		int destoryCudaStream(stream_t stream)
		{
			return hipStreamDestroy((hipStream_t)stream);
		}

		int asyncCudaStream(stream_t stream)
		{
			return hipStreamSynchronize((hipStream_t)stream);
		}

		const char* getCudaError(int errorId)
		{
			return hipGetErrorString(static_cast<hipError_t>(errorId));
		}

		int getLastError()
		{
			return hipGetLastError();
		}
	}
}
